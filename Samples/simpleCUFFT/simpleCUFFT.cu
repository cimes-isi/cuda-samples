#include "hip/hip_runtime.h"
/* Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/* Example showing the use of CUFFT for fast 1D-convolution using FFT. */

// includes, system
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

// Complex data type
typedef float2 Complex;
static __device__ __host__ inline Complex ComplexAdd(Complex, Complex);
static __device__ __host__ inline Complex ComplexScale(Complex, float);
static __device__ __host__ inline Complex ComplexMul(Complex, Complex);
static __global__ void ComplexPointwiseMulAndScale(Complex *, const Complex *,
                                                   int, float);

// Filtering functions
void Convolve(const Complex *, int, const Complex *, int, Complex *);

// Padding functions
int PadData(const Complex *, Complex **, int, const Complex *, Complex **, int);

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);

#define DO_CHECK_RESULT 1

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) { runTest(argc, argv); }

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char **argv) {
  printf("[simpleCUFFT] is starting...\n");

  int signal_size = 64;
  if (argc > 1) {
    signal_size = atoi(argv[1]);
  }
// The filter size is assumed to be a number smaller than the signal size
  int filter_kernel_size = signal_size / 4;
  if (argc > 2) {
    filter_kernel_size = atoi(argv[2]);
  }
  printf("Signal size = %zu\n", signal_size);
  printf("Filter size = %zu\n", filter_kernel_size);

  findCudaDevice(argc, (const char **)argv);

  // Allocate host memory for the signal
  Complex *h_signal =
      reinterpret_cast<Complex *>(malloc(sizeof(Complex) * signal_size));

  // Initialize the memory for the signal
  for (unsigned int i = 0; i < signal_size; ++i) {
    h_signal[i].x = rand() / static_cast<float>(RAND_MAX);
    h_signal[i].y = 0;
  }

  // Allocate host memory for the filter
  Complex *h_filter_kernel =
      reinterpret_cast<Complex *>(malloc(sizeof(Complex) * filter_kernel_size));

  // Initialize the memory for the filter
  for (unsigned int i = 0; i < filter_kernel_size; ++i) {
    h_filter_kernel[i].x = rand() / static_cast<float>(RAND_MAX);
    h_filter_kernel[i].y = 0;
  }

  // Pad signal and filter kernel
  Complex *h_padded_signal;
  Complex *h_padded_filter_kernel;
  int new_size =
      PadData(h_signal, &h_padded_signal, signal_size, h_filter_kernel,
              &h_padded_filter_kernel, filter_kernel_size);
  int mem_size = sizeof(Complex) * new_size;

  // Allocate CUDA events that we'll use for timing
  hipEvent_t start;
  checkCudaErrors(hipEventCreate(&start));

  hipEvent_t stop;
  checkCudaErrors(hipEventCreate(&stop));

  hipEvent_t start_kernel;
  checkCudaErrors(hipEventCreate(&start_kernel));

  hipEvent_t stop_kernel;
  checkCudaErrors(hipEventCreate(&stop_kernel));

  // Record the start event
  checkCudaErrors(hipEventRecord(start, NULL));

  // Allocate device memory for signal
  Complex *d_signal;
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_signal), mem_size));
  // Copy host memory to device
  checkCudaErrors(
      hipMemcpy(d_signal, h_padded_signal, mem_size, hipMemcpyHostToDevice));

  // Allocate device memory for filter kernel
  Complex *d_filter_kernel;
  checkCudaErrors(
      hipMalloc(reinterpret_cast<void **>(&d_filter_kernel), mem_size));

  // Copy host memory to device
  checkCudaErrors(hipMemcpy(d_filter_kernel, h_padded_filter_kernel, mem_size,
                             hipMemcpyHostToDevice));

  // CUFFT plan simple API
  hipfftHandle plan;
  checkCudaErrors(hipfftPlan1d(&plan, new_size, HIPFFT_C2C, 1));

  // CUFFT plan advanced API
  hipfftHandle plan_adv;
  size_t workSize;
  long long int new_size_long = new_size;

  checkCudaErrors(hipfftCreate(&plan_adv));
  checkCudaErrors(hipfftXtMakePlanMany(plan_adv, 1, &new_size_long, NULL, 1, 1,
                                      HIP_C_32F, NULL, 1, 1, HIP_C_32F, 1,
                                      &workSize, HIP_C_32F));
  printf("Temporary buffer size %li bytes\n", workSize);

  // Record the start_kernel event
  checkCudaErrors(hipEventRecord(start_kernel, NULL));

  // Transform signal and kernel
  printf("Transforming signal hipfftExecC2C\n");
  checkCudaErrors(hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(d_signal),
                               reinterpret_cast<hipfftComplex *>(d_signal),
                               HIPFFT_FORWARD));
  checkCudaErrors(hipfftExecC2C(
      plan_adv, reinterpret_cast<hipfftComplex *>(d_filter_kernel),
      reinterpret_cast<hipfftComplex *>(d_filter_kernel), HIPFFT_FORWARD));

  // Multiply the coefficients together and normalize the result
  printf("Launching ComplexPointwiseMulAndScale<<< >>>\n");
  ComplexPointwiseMulAndScale<<<32, 256>>>(d_signal, d_filter_kernel, new_size,
                                           1.0f / new_size);

  // Check if kernel execution generated and error
  getLastCudaError("Kernel execution failed [ ComplexPointwiseMulAndScale ]");

  // Transform signal back
  printf("Transforming signal back hipfftExecC2C\n");
  checkCudaErrors(hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(d_signal),
                               reinterpret_cast<hipfftComplex *>(d_signal),
                               HIPFFT_BACKWARD));

  // Record the stop_kernel event
  checkCudaErrors(hipEventRecord(stop_kernel, NULL));

  // Copy device memory to host
  Complex *h_convolved_signal = h_padded_signal;
  checkCudaErrors(hipMemcpy(h_convolved_signal, d_signal, mem_size,
                             hipMemcpyDeviceToHost));

  // Record the stop event
  checkCudaErrors(hipEventRecord(stop, NULL));

  // Wait for the stop_kernel event to complete
  checkCudaErrors(hipEventSynchronize(stop_kernel));
  // Wait for the stop event to complete
  checkCudaErrors(hipEventSynchronize(stop));

  float msecKernel = 0.0f;
  checkCudaErrors(hipEventElapsedTime(&msecKernel, start_kernel, stop_kernel));
  float msecTotal = 0.0f;
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

  // Compute and print the performance
  printf("Kernel Time = %.3f msec\n", msecKernel);
  printf("Total Time  = %.3f msec\n", msecTotal);

  bool bTestResult = true;
#if DO_CHECK_RESULT
  // Allocate host memory for the convolution result
  Complex *h_convolved_signal_ref =
      reinterpret_cast<Complex *>(malloc(sizeof(Complex) * signal_size));

  // Convolve on the host
  Convolve(h_signal, signal_size, h_filter_kernel, filter_kernel_size,
           h_convolved_signal_ref);

  // check result
  bTestResult = sdkCompareL2fe(
      reinterpret_cast<float *>(h_convolved_signal_ref),
      reinterpret_cast<float *>(h_convolved_signal), 2 * signal_size, 1e-5f);
  free(h_convolved_signal_ref);
#endif

  // Destroy CUFFT context
  checkCudaErrors(hipfftDestroy(plan));
  checkCudaErrors(hipfftDestroy(plan_adv));

  // cleanup memory
  free(h_signal);
  free(h_filter_kernel);
  free(h_padded_signal);
  free(h_padded_filter_kernel);
  checkCudaErrors(hipFree(d_signal));
  checkCudaErrors(hipFree(d_filter_kernel));

  exit(bTestResult ? EXIT_SUCCESS : EXIT_FAILURE);
}

// Pad data
int PadData(const Complex *signal, Complex **padded_signal, int signal_size,
            const Complex *filter_kernel, Complex **padded_filter_kernel,
            int filter_kernel_size) {
  int minRadius = filter_kernel_size / 2;
  int maxRadius = filter_kernel_size - minRadius;
  int new_size = signal_size + maxRadius;

  // Pad signal
  Complex *new_data =
      reinterpret_cast<Complex *>(malloc(sizeof(Complex) * new_size));
  memcpy(new_data + 0, signal, signal_size * sizeof(Complex));
  memset(new_data + signal_size, 0, (new_size - signal_size) * sizeof(Complex));
  *padded_signal = new_data;

  // Pad filter
  new_data = reinterpret_cast<Complex *>(malloc(sizeof(Complex) * new_size));
  memcpy(new_data + 0, filter_kernel + minRadius, maxRadius * sizeof(Complex));
  memset(new_data + maxRadius, 0,
         (new_size - filter_kernel_size) * sizeof(Complex));
  memcpy(new_data + new_size - minRadius, filter_kernel,
         minRadius * sizeof(Complex));
  *padded_filter_kernel = new_data;

  return new_size;
}

////////////////////////////////////////////////////////////////////////////////
// Filtering operations
////////////////////////////////////////////////////////////////////////////////

// Computes convolution on the host
void Convolve(const Complex *signal, int signal_size,
              const Complex *filter_kernel, int filter_kernel_size,
              Complex *filtered_signal) {
  int minRadius = filter_kernel_size / 2;
  int maxRadius = filter_kernel_size - minRadius;

  // Loop over output element indices
  for (int i = 0; i < signal_size; ++i) {
    filtered_signal[i].x = filtered_signal[i].y = 0;

    // Loop over convolution indices
    for (int j = -maxRadius + 1; j <= minRadius; ++j) {
      int k = i + j;

      if (k >= 0 && k < signal_size) {
        filtered_signal[i] =
            ComplexAdd(filtered_signal[i],
                       ComplexMul(signal[k], filter_kernel[minRadius - j]));
      }
    }
  }
}

////////////////////////////////////////////////////////////////////////////////
// Complex operations
////////////////////////////////////////////////////////////////////////////////

// Complex addition
static __device__ __host__ inline Complex ComplexAdd(Complex a, Complex b) {
  Complex c;
  c.x = a.x + b.x;
  c.y = a.y + b.y;
  return c;
}

// Complex scale
static __device__ __host__ inline Complex ComplexScale(Complex a, float s) {
  Complex c;
  c.x = s * a.x;
  c.y = s * a.y;
  return c;
}

// Complex multiplication
static __device__ __host__ inline Complex ComplexMul(Complex a, Complex b) {
  Complex c;
  c.x = a.x * b.x - a.y * b.y;
  c.y = a.x * b.y + a.y * b.x;
  return c;
}

// Complex pointwise multiplication
static __global__ void ComplexPointwiseMulAndScale(Complex *a, const Complex *b,
                                                   int size, float scale) {
  const int numThreads = blockDim.x * gridDim.x;
  const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

  for (int i = threadID; i < size; i += numThreads) {
    a[i] = ComplexScale(ComplexMul(a[i], b[i]), scale);
  }
}
